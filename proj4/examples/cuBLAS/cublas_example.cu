/*
 * Reference:
 * https://developer.nvidia.com/sites/default/files/akamai/cuda/files/Misc/mygpu.pdf
 */

# include <stdio.h>
# include <stdlib.h>
# include <hip/hip_runtime.h>
# include "hipblas.h"

# define IDX2C(i,j,ld) ((( j )*( ld ))+( i ))
# define m 6
# define n 4 
# define k 5 
int main()
{
	hipError_t cudaStat; 
	hipblasStatus_t stat; 
	hipblasHandle_t handle; 
	int i, j; 
	float *a; 
	float *b; 
	float *c; 
	a = (float *)malloc(m * k * sizeof(float)); 
	b = (float *)malloc(k * n * sizeof(float)); 
	c = (float *)malloc(m * n * sizeof(float)); 
	
	int ind = 11; 
	for(j = 0; j < k; j++){ 
		for(i = 0; i < m; i++){ 
			a[IDX2C(i, j, m)] = (float)ind++; 
		} 
	} 
	
	printf ("a:\n");
	for (i=0;i<m;i ++){
		for (j=0;j<k;j ++){
			printf (" %5.0f", a[ IDX2C(i,j,m )]);
		}
		printf ("\n");
	}
	
	ind =11; 
	for(j=0;j<n;j ++){ 
		for(i=0;i<k;i ++){ 
			b[ IDX2C(i,j,k )]=( float )ind ++; 
		} 
	} 
	
	printf ("b:\n");
	for (i=0;i<k;i ++){
		for (j=0;j<n;j ++){
			printf (" %5.0f",b[ IDX2C(i, j, k)]);
		}
		printf ("\n");
	}
	
	ind =11; 
	for(j=0;j<n;j ++){ 
		for(i=0;i<m;i ++){ 
			c[ IDX2C(i,j,m )]=( float )ind ++; 
		} 
	} 
	
	
	printf ("c:\n");
	for (i=0;i<m;i ++){
		for (j=0;j<n;j ++){
			printf (" %5.0f",c[ IDX2C(i,j,m )]);
		}
		printf ("\n");
	}
	
	float * d_a; 
	float * d_b; 
	float * d_c; 
	cudaStat = hipMalloc (( void **)& d_a ,m*k* sizeof (*a)); 
	
	cudaStat = hipMalloc (( void **)& d_b ,k*n* sizeof (*b)); 
	
	cudaStat = hipMalloc (( void **)& d_c ,m*n* sizeof (*c)); 
	
	stat = hipblasCreate (& handle ); 
	
	stat = hipblasSetMatrix (m,k, sizeof (*a) ,a,m,d_a ,m); 
	stat = hipblasSetMatrix (k,n, sizeof (*b) ,b,k,d_b ,k); 
	stat = hipblasSetMatrix (m,n, sizeof (*c) ,c,m,d_c ,m); 

	float al =1.0f; 
	float bet =1.0f; 
	
	stat = hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,n,k,&al,d_a,m,d_b,k,&bet,d_c,m);
	stat = hipblasGetMatrix (m,n, sizeof (*c) ,d_c ,m,c,m); 
	printf ("c after Sgemm :\n");
	for(i=0;i<m;i ++){
		for(j=0;j<n;j ++){
			printf (" %7.0f",c[ IDX2C(i, j, m )]); 
		}
		printf ("\n");
	}
	hipFree (d_a ); 
	hipFree (d_b ); 
	hipFree (d_c ); 
	hipblasDestroy ( handle ); 
	free (a); 
	free (b); 
	free (c); 
	return EXIT_SUCCESS ;
}
