/*
 * Reference:
 * https://developer.nvidia.com/sites/default/files/akamai/cuda/files/Misc/mygpu.pdf
 */

#include "hipblas.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define IDX2C(i, j, ld) (((j) * (ld)) + (i))
#define m 32
#define n 32
#define k 32

#include <stddef.h>
#include <sys/resource.h>
#include <sys/sysinfo.h>
#include <sys/time.h>
static double second(void) {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
}

typedef __half half1;

__global__ void half2float(float *dout, half *din) {
    int i = threadIdx.x;
    dout[i] = __half2float(din[i]);
}

__global__ void float2half(half *dout, float *din) {
    int i = threadIdx.x;
    dout[i] = __float2half(din[i]);
}

int main(int argc, char *argv[]) {
  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  int i, j;
  float *a;
  float *b;
  float *c;
  double start, end;
  hipblasMath_t mode;

  if (argc != 2) {
    fprintf(stderr, "Usage: %s <ON|OFF>\n", argv[0]);
    fprintf(stderr, "\tSpecify whether you want to turn on the Tensor Cores\n");
    exit(1);
  }
  if (!strcmp(argv[1], "ON")) {
    printf("Enabling Tensor Core\n");
    mode = HIPBLAS_TENSOR_OP_MATH;
  } else if (!strcmp(argv[1], "OFF")) {
    printf("Disabling Tensor Core\n");
    mode = HIPBLAS_DEFAULT_MATH;
  } else {
    fprintf(stderr, "Invalid argument\n");
    exit(1);
  }

  a = (float *)malloc(m * k * sizeof(float));
  b = (float *)malloc(k * n * sizeof(float));
  c = (float *)malloc(m * n * sizeof(float));

  float ind = 0.001;
  for (j = 0; j < k; j++) {
    for (i = 0; i < m; i++) {
      a[IDX2C(i, j, m)] = ind*j;
    }
  }

  ind = 0.0001;
  for (j = 0; j < n; j++) {
    for (i = 0; i < k; i++) {
      b[IDX2C(i, j, k)] = ind*j;
    }
  }

  float *d_a;
  half1 *d_a_half;
  float *d_b;
  half1 *d_b_half;
  float *d_c;
  half1 *d_c_half;

  cudaStat = hipMalloc((void **)&d_a, m * k * sizeof(*a));
  cudaStat = hipMalloc((void **)&d_b, k * n * sizeof(*b));
  cudaStat = hipMalloc((void **)&d_c, m * n * sizeof(*c));

  cudaStat = hipMalloc((void **)&d_a_half, m * k * sizeof(*a));
  cudaStat = hipMalloc((void **)&d_b_half, k * n * sizeof(*b));
  cudaStat = hipMalloc((void **)&d_c_half, m * n * sizeof(*c));

  stat = hipblasCreate(&handle);

  hipblasSetMathMode(handle, mode);

  stat = hipblasSetMatrix(m, k, sizeof(*a), a, m, d_a, m);
  stat = hipblasSetMatrix(k, n, sizeof(*b), b, k, d_b, k);
  stat = hipblasSetMatrix(m, n, sizeof(*c), c, m, d_c, m);

  
  float2half<<<1, m*k>>>(d_a_half, d_a);
  float2half<<<1, k*n>>>(d_b_half, d_b);

  half1 al = 1.0f;
  half1 bet = 1.0f;

  start = second();
  for (int i = 0; i < 100; i++) {
    if ((stat = hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &al, 
                    d_a_half, m, d_b_half, k, &bet, d_c_half, m)) !=
        HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "Hgemm failed\n");
      exit(1);
    }
  }
  end = second();

  half2float<<<1, m*n>>>(d_c, d_c_half);

  stat = hipblasGetMatrix(m, n, sizeof(*c), d_c, m, c, m);
  printf("c after Sgemm :\n");
  for (i = 0; i < m; i++) {
    for (j = 0; j < n; j++) {
      printf(" %.2f", __half2float(c[IDX2C(i, j, m)]));
    }
    printf("\n");
  }
  printf("Time took: %lf\n", end - start);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  hipblasDestroy(handle);
  free(a);
  free(b);
  free(c);
  return EXIT_SUCCESS;
}
