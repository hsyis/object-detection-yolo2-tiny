# include <hip/hip_runtime.h>
# include "hipblas.h"

extern "C"
void cublas_mul_float(float *a, float *b, float *c, int m, int k, int n)
{
	hipblasHandle_t handle; 
	
	float * d_a; 
	float * d_b; 
	float * d_c; 
	hipMalloc (( void **)& d_a ,m*k* sizeof (*a)); 
	
	hipMalloc (( void **)& d_b ,k*n* sizeof (*b)); 
	
	hipMalloc (( void **)& d_c ,m*n* sizeof (*c)); 
	
	hipblasCreate (& handle ); 
	
	hipblasSetMatrix (m,k, sizeof (*a) ,a,m,d_a ,m); 
	hipblasSetMatrix (k,n, sizeof (*b) ,b,k,d_b ,k); 
	hipblasSetMatrix (m,n, sizeof (*c) ,c,m,d_c ,m); 

	float al =1.0f; 
	float bet =0.0f; 

    // column-major    
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        m, n, k,
        &al,
        d_a, m,
        d_b, k,
        &bet,
        d_c, m
    );
	hipblasGetMatrix (m,n, sizeof (*c) ,d_c ,m,c,m); 

	hipFree (d_a ); 
	hipFree (d_b ); 
	hipFree (d_c ); 
	hipblasDestroy ( handle ); 
}
