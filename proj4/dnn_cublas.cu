#include "hip/hip_runtime.h"
# include <hip/hip_runtime.h>
# include "hipblas.h"

extern "C"
void cublas_mul_float(float *a, float *b, float *c, int m, int k, int n)
{
	hipblasHandle_t handle; 
	
	float * d_a; 
	float * d_b; 
	float * d_c; 
	hipMalloc (( void **)& d_a ,m*k* sizeof (*a)); 
	
	hipMalloc (( void **)& d_b ,k*n* sizeof (*b)); 
	
	hipMalloc (( void **)& d_c ,m*n* sizeof (*c)); 
	
	hipblasCreate (& handle ); 
	
	hipblasSetMatrix (m,k, sizeof (*a) ,a,m,d_a ,m); 
	hipblasSetMatrix (k,n, sizeof (*b) ,b,k,d_b ,k); 
	hipblasSetMatrix (m,n, sizeof (*c) ,c,m,d_c ,m); 

	float al =1.0f; 
	float bet =0.0f; 

    // column-major    
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        m, n, k,
        &al,
        d_a, m,
        d_b, k,
        &bet,
        d_c, m
    );
	hipblasGetMatrix (m,n, sizeof (*c) ,d_c ,m,c,m); 

	hipFree (d_a ); 
	hipFree (d_b ); 
	hipFree (d_c ); 
	hipblasDestroy ( handle ); 
}

extern "C"
void cublas_max_pool_float(float *a, float *c, int m1, int m2, int m3)
{
    for (int i = 0; i < m1; i++) {
        for (int j = 0; j < m3; j++) {
            float max = a[i * m2 * m3 + j];
            for (int k = 1; k < m2; k++) {
                float tmp = a[i * m2 * m3 + k * m3 + j];
                if (tmp > max) {
                    max = tmp;
                }

            }
            c[i * m3 + j] = max;
        }
    }
}

extern "C"
void cublas_norm_float(float *a, float *c, int m, int k, float *gamma, float *mean, float* variance, float epsilon)
{
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            c[i * k + j] = gamma[j] * (a[i * k + j] - mean[j]) / sqrt(variance[j] + epsilon);
        }
    }
}

extern "C"
void cublas_leaky_relu_float(float *a, float *c, int m)
{
    for (int i = 0; i < m; i++) {
        c[i] = max(0.1 * a[i], a[i]);
    }
}
